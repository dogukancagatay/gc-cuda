#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <stdio.h>

#include <boost/unordered_map.hpp>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "api/utility.hpp"
#include "api/graph.hpp"

#define threadsPerBlock 32
#define RANDOMRESETPROB 0.15

using namespace std;

__global__ void vertexUpdate(int iteration, int activeShardLength, float *activeShardContent_d, int firstActiveVertex, int firstActiveVertexInNextShard, float *result_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int assignedVertexId = idx + firstActiveVertex;
	
	float src, dest, value;
	float pagerank = 1, sum;

	if(assignedVertexId<firstActiveVertexInNextShard) {
		if(iteration != 0) {
			sum = 0;
			
			for(int i=0; i<activeShardLength; i++) {
				src = activeShardContent_d[i*3+0];
				dest = activeShardContent_d[i*3+1];
				value = activeShardContent_d[i*3+2];

				if(dest == assignedVertexId) {
					sum = sum + value;
				}
			}

			pagerank = RANDOMRESETPROB + (1 - RANDOMRESETPROB) * sum;
		} 

		result_d[idx] = pagerank/result_d[idx];
	}
}

int getActiveShardLength(int firstActiveVertex, int firstActiveVertexInNextShard, graph_t *graph) {
	int total = 0;

	for(int i=firstActiveVertex; i<firstActiveVertexInNextShard; i++)
		total = total + graph->in_edge_counts[i];

	return total;
}

void gc_cuda(params *par, graph_t *graph){

	cout<<endl<<endl<<"gc_cuda"<<endl;
	int num_iterations = par->num_iterations;

    FILE **ifShards = (FILE **)malloc(sizeof(FILE *) * par->num_shards);
    FILE **tfShards = (FILE **)malloc(sizeof(FILE *) * par->num_shards);
    int *srcs = (int *)malloc(sizeof(int) * par->num_shards);
    
    for(int i=0; i<num_iterations; i++) {
    	cout << "------------iteration: "<<i<<"------------"<<endl<<endl;
    	// open shards to read
    	for(int j=0; j<par->num_shards; j++) {
    		ifShards[j] = fopen(graph->shard_fnames[j].c_str(), "r");
    	}

    	// open temp shards to write
    	for(int j=0; j<par->num_shards; j++) {
    		tfShards[j] = fopen(("temp"+graph->shard_fnames[j]).c_str(), "w");
    	}

    	// init srcs with -1
    	for(int j=0; j<par->num_shards; j++) {
    		srcs[j] = -1;
    	}

    	for(int activeShard=0; activeShard<par->num_shards; activeShard++){
    		FILE *ifActiveShard = fopen(graph->shard_fnames[activeShard].c_str(), "r");

    		int firstActiveVertex = graph->shard_to_node[activeShard];
    		int firstActiveVertexInNextShard = graph->shard_to_node[activeShard+1];
			int activeShardLength = getActiveShardLength(firstActiveVertex, firstActiveVertexInNextShard, graph);

			// memory alloc on host and device for active shard content
			float *activeShardContent;
			float *activeShardContent_d;

			activeShardContent = (float *)malloc(activeShardLength * 3 * sizeof(float));
			hipMalloc((void **) &activeShardContent_d, activeShardLength * 3 * sizeof(float));
						
			// memory alloc on host and device for kernel result
			// result initially stores out edge counts
			float *result;
			float *result_d;

			result = (float *)malloc((firstActiveVertexInNextShard - firstActiveVertex) * sizeof(float));
			for(int k=0; k<(firstActiveVertexInNextShard - firstActiveVertex); k++)
				result[k] = graph->out_edge_counts[k+firstActiveVertex];
			hipMalloc((void **) &result_d, (firstActiveVertexInNextShard - firstActiveVertex) * sizeof(float));

			// read active shard content
			for(int k=0; k<activeShardLength*3; k++){
				fscanf(ifActiveShard,"%f ", &activeShardContent[k]);
			}

			// copy active shard from host to device
			hipMemcpy(activeShardContent_d, activeShardContent, activeShardLength * 3 * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(result_d, result, (firstActiveVertexInNextShard - firstActiveVertex) * sizeof(float), hipMemcpyHostToDevice);

			// vertex update
			int blocksPerGrid = (firstActiveVertexInNextShard - firstActiveVertex + threadsPerBlock -1)/threadsPerBlock;
			vertexUpdate<<<blocksPerGrid,threadsPerBlock>>>(i, activeShardLength, activeShardContent_d, firstActiveVertex, firstActiveVertexInNextShard, result_d);

			// copy output from device to host
			hipMemcpy(result, result_d, (firstActiveVertexInNextShard - firstActiveVertex) * sizeof(float), hipMemcpyDeviceToHost);

			// write output to temp shard
			int src, dest;
			float value;
			cout<<"activeShard: "<<activeShard<<endl;
			for(int k=0; k<par->num_shards; k++) {
				while(true) {
					if(feof(ifShards[k]))
						break;

					if(srcs[k] == -1) {
						fscanf(ifShards[k], "%d ",&src);
					}
					else {
						src = srcs[k];
						srcs[k] = -1;
					}

					if(src < firstActiveVertexInNextShard) {
						fscanf(ifShards[k], "%d %f ",&dest, &value);
						fprintf(tfShards[k], "%d %d %f\n", src, dest, result[src - firstActiveVertex]);
						//cout<<"update shard: "<<k<<" ---- src: "<<src<<" dest: "<<dest<<" value: "<<result[src - firstActiveVertex]<<endl;
					}
					else {
						srcs[k] = src;
						break;
					}
				}
			}

			// write pagerank result to stdout
			for(int i=0; i<(firstActiveVertexInNextShard - firstActiveVertex); i++) {
				int vertexID = i + firstActiveVertex;
				cout <<"vertex id: "<<vertexID<< " pagerank: "<<result[i] * graph->out_edge_counts[vertexID]<<endl;
			}

			// dealloc active shard content on host and device
			delete[] activeShardContent;
			activeShardContent = 0;
			hipFree(activeShardContent_d);

			// dealloc result on host and device
			delete[] result;
			result = 0;
			hipFree(result_d);

			fclose(ifActiveShard);
    	}
    	
    	// close files
    	for(int j=0; j<par->num_shards; j++) {
    		fclose(ifShards[j]);
    		fclose(tfShards[j]);
    	}

    	// rename temp files for the next iteration
    	for(int j=0; j<par->num_shards; j++) {
    		rename( ("temp"+graph->shard_fnames[j]).c_str(), (graph->shard_fnames[j]).c_str());
    	}
    }
    cout<<"end of cuda"<<endl;
}
