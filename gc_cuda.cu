#include <iostream>
#include <vector>
#include <string>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "api/utility.hpp"
#include "api/graph.hpp"

using namespace std;

void gc_cuda(params *par, graph_t *graph){
    
    // number of shards, filenames of shards
    for(int i=0;i<par->num_shards; i++) {
        std::cout<<graph->shard_fnames[i]<< "\n";
    }
}


void deneme(){
}
